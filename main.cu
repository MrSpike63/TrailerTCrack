#include <cinttypes>
#include <iostream>
#include <chrono>

#include <hip/hip_runtime.h>


#define THREAD_COUNT 128
#define TASK_WORK (1ULL << 30)

#define FAST_NEXT_INT


#ifdef BOINC
    #include "boinc_api.h"
    #if defined _WIN32 || defined _WIN64
        #include "boinc_win.h"
    #endif
#endif

#ifndef BOINC
    #define boinc_fopen(file, mode) fopen(file, mode)
    #define boinc_delete_file(file) remove(file)

    #define boinc_begin_critical_section()
    #define boinc_end_critical_section()

    #define boinc_fraction_done(frac)
    #define boinc_finish(s) exit(s)

    #define boinc_time_to_checkpoint() true
    #define boinc_checkpoint_completed()
#endif



namespace Random {
    #define RANDOM__MULTIPLIER 25214903917ULL
    #define RANDOM__MULTIPLIER_INVERSE 246154705703781ULL

    #define RANDOM__ADDEND 11ULL
    #define RANDOM__ADDEND_INVERSE 107048004364969ULL
    #define RANDOM__MASK ((1ULL << 48) - 1)

    __device__ uint64_t setSeed(uint64_t seed) {
        return (seed ^ RANDOM__MULTIPLIER) & RANDOM__MASK;
    }

    __device__ int32_t next(uint64_t &seed, int bits) {
        seed = (seed * RANDOM__MULTIPLIER + RANDOM__ADDEND) & RANDOM__MASK;

        return (int32_t)(seed >> (48 - bits));
    }

    __device__ int32_t nextInt(uint64_t &seed) {
        return next(seed, 32);
    }

    __device__ int32_t nextInt(uint64_t &seed, int bound) {
        if ((bound & -bound) == bound) {
            seed = (seed * RANDOM__MULTIPLIER + RANDOM__ADDEND) & RANDOM__MASK;
            return (int32_t)((bound * (seed >> 17)) >> 31);
        }
    
        int32_t bits, value;
        #ifndef FAST_NEXT_INT
        do {
        #endif
            seed = (seed * RANDOM__MULTIPLIER + RANDOM__ADDEND) & RANDOM__MASK;
            bits = seed >> 17;
            value = bits % bound;
        
        #ifndef FAST_NEXT_INT
        } while (bits - value + (bound - 1) < 0);
        #endif
        return value;
    }

    __device__ uint64_t nextLong(uint64_t &seed) {
        return ((uint64_t)next(seed, 32) << 32) + next(seed, 32);
    }

    __device__ float nextFloat(uint64_t &seed) {
        return next(seed, 24) / ((float)(1 << 24));
    }

    __device__ double nextDouble(uint64_t &seed) {
        return (((uint64_t)next(seed, 26) << 27) + next(seed, 27)) / (double)(1ULL << 53);
    }

    template <int n>
    __device__ constexpr void advance(uint64_t &seed) {
        uint64_t m = 1;
        uint64_t a = 0;
        for (int i = 0; i < n; i++) {
            a = (a * RANDOM__MULTIPLIER + RANDOM__ADDEND) & RANDOM__MASK;
            m = (m * RANDOM__MULTIPLIER) & RANDOM__MASK;
        }

        seed = (seed * m + a) & RANDOM__MASK;
    }
}


__shared__ uint8_t sharedMemory[256 * THREAD_COUNT];
#define SHARED_MEMORY_ACCESS(n) sharedMemory[(threadIdx.x << 8) | n]
#define CASTED_SHARED_MEMORY_ACCESS(n) ((double*)sharedMemory)[(threadIdx.x << 5) | n]


namespace Terrain {
    struct OctaveData {
        double xOffset;
        double yOffset;
        double zOffset;
        uint8_t permutations[256];
    };

    struct NoiseData {
        OctaveData noise1[16];
        OctaveData noise2[16];
        OctaveData noise3[8];
        OctaveData noise6[16];
    };

    __device__ void initializeOctave(uint64_t &random, OctaveData *octaveData) {
        octaveData->xOffset = Random::nextDouble(random) * 256.0;
        octaveData->yOffset = Random::nextDouble(random) * 256.0;
        octaveData->zOffset = Random::nextDouble(random) * 256.0;


        for (int i = 0; i < 256; i++) {
            SHARED_MEMORY_ACCESS(i) = i;
        }

        for (int i = 0; i < 256; i++) {
            uint8_t k = Random::nextInt(random, 256 - i) + i;
            uint8_t l = SHARED_MEMORY_ACCESS(i);
            octaveData->permutations[i] = SHARED_MEMORY_ACCESS(k);
            SHARED_MEMORY_ACCESS(k) = l;
        }
    }

    __device__ void initializeNoise(uint64_t worldSeed, NoiseData* noiseData) {
        uint64_t random = Random::setSeed(worldSeed);

        for (int i = 0; i < 16; i++) { initializeOctave(random, &noiseData->noise1[i]); }
        for (int i = 0; i < 16; i++) { initializeOctave(random, &noiseData->noise2[i]); }
        for (int i = 0; i < 8; i++) { initializeOctave(random, &noiseData->noise3[i]); }


        #ifndef FAST_NEXT_INT
            for (int i = 0; i < 14; i++) {
                Random::advance<7>(random);
                for (int j = 1; j < 256; j++) {
                    Random::nextInt(random, 256 - j);
                }
            }
        #else
            Random::advance<3668>(random);
        #endif

        for (int i = 0; i < 16; i++) { initializeOctave(random, &noiseData->noise6[i]); }
    }



    __device__ double lerp(double t, double a, double b) {
        return a + t * (b - a);
    }



    __device__ double func_4110_a(int i, double x, double z) {
        switch (i & 0xF) {
            case 0x0:
                return x;
            case 0x1:
                return -x;
            case 0x2:
                return x;
            case 0x3:
                return -x;
            case 0x4:
                return x + z;
            case 0x5:
                return -x + z;
            case 0x6:
                return x - z;
            case 0x7:
                return -x - z;
            case 0x8:
                return z;
            case 0x9:
                return -z;
            case 0xA:
                return -z;
            case 0xB:
                return -z;
            case 0xC:
                return x;
            case 0xD:
                return z;
            case 0xE:
                return -x;
            case 0xF:
                return -z;
            default:
                return 0;
        }
    }


    __device__ double grad(int i, double x, double y, double z) {
        switch (i & 0xF) {
            case 0x0:
                return x + y;
            case 0x1:
                return -x + y;
            case 0x2:
                return x - y;
            case 0x3:
                return -x - y;
            case 0x4:
                return x + z;
            case 0x5:
                return -x + z;
            case 0x6:
                return x - z;
            case 0x7:
                return -x - z;
            case 0x8:
                return y + z;
            case 0x9:
                return -y + z;
            case 0xA:
                return y - z;
            case 0xB:
                return -y - z;
            case 0xC:
                return y + x;
            case 0xD:
                return -y + z;
            case 0xE:
                return y - x;
            case 0xF:
                return -y - z;
            default:
                return 0;
        }
    }


    __device__ uint8_t getPermutation(const uint8_t* __restrict__ permutations, int n) {
        return permutations[n & 0xFF];
    }


    __device__ double optimizedNoise2D(const OctaveData* __restrict__ octaveDatas, double baseX, double baseZ, int xIteration, int zIteration, double noiseScaleX, double noiseScaleZ, int numOctaves) {
        double outputValue = 0;

        double octavesFactor = 1.0;
        for (int i = 0; i < numOctaves; i++) {
            double noiseFactorX = noiseScaleX * octavesFactor;
            double noiseFactorZ = noiseScaleZ * octavesFactor;

            double startX = (double)baseX * octavesFactor * noiseScaleX;
            double startZ = (double)baseZ * octavesFactor * noiseScaleZ;

            double octaveWidth = 1.0 / octavesFactor;

            double xCoord = startX + (double)xIteration * noiseFactorX + octaveDatas[i].xOffset;
            int xCoordFloor = (int)xCoord;
            if (xCoord < (double)xCoordFloor) {
                xCoordFloor--;
            }
            int xUnitCube = xCoordFloor & 0xFF;
            xCoord -= xCoordFloor;
            double fadeX = xCoord * xCoord * xCoord * (xCoord * (xCoord * 6.0 - 15.0) + 10.0);

            double zCoord = startZ + (double)zIteration * noiseFactorZ + octaveDatas[i].zOffset;
            int zCoordFloor = (int)zCoord;
            if (zCoord < (double)zCoordFloor) {
                zCoordFloor--;
            }
            int zUnitCube = zCoordFloor & 0xFF;
            zCoord -= zCoordFloor;
            double fadeZ = zCoord * zCoord * zCoord * (zCoord * (zCoord * 6.0 - 15.0) + 10.0);

            int l = getPermutation(octaveDatas[i].permutations, xUnitCube) + 0;
            int j1 = getPermutation(octaveDatas[i].permutations, l) + zUnitCube;
            int k1 = getPermutation(octaveDatas[i].permutations, xUnitCube + 1) + 0;
            int l1 = getPermutation(octaveDatas[i].permutations, k1) + zUnitCube;

            double d9 = lerp(fadeX, func_4110_a(getPermutation(octaveDatas[i].permutations, j1), xCoord, zCoord), grad(getPermutation(octaveDatas[i].permutations, l1), xCoord - 1.0, 0.0, zCoord));
            double d11 = lerp(fadeX, grad(getPermutation(octaveDatas[i].permutations, j1 + 1), xCoord, 0.0, zCoord - 1.0), grad(getPermutation(octaveDatas[i].permutations, l1 + 1), xCoord - 1.0, 0.0, zCoord - 1.0));
            double d23 = lerp(fadeZ, d9, d11);
            outputValue += d23 * octaveWidth;


            octavesFactor /= 2.0;
        }

        return outputValue;
    }

    __device__ void optimizedNoise3D(const OctaveData* __restrict__ octaveDatas, int sharedMemoryOffset, double baseX, double baseY, double baseZ, int xIteration, int zIteration, double noiseScaleX, double noiseScaleY, double noiseScaleZ, int numOctaves, int yIterationStart, int yIterations) {
        double octavesFactor = 1.0;
        for (int i = 0; i < numOctaves; i++) {
            double noiseFactorX = noiseScaleX * octavesFactor;
            double noiseFactorY = noiseScaleY * octavesFactor;
            double noiseFactorZ = noiseScaleZ * octavesFactor;

            double startX = (double)baseX * octavesFactor * noiseScaleX;
            double startY = (double)baseY * octavesFactor * noiseScaleY;
            double startZ = (double)baseZ * octavesFactor * noiseScaleZ;

            int i2 = -1;
            double d13 = 0.0;
            double d15 = 0.0;
            double d16 = 0.0;
            double d18 = 0.0;

            double octaveWidth = 1.0 / octavesFactor;

            double xCoord = startX + (double)xIteration * noiseFactorX + octaveDatas[i].xOffset;
            int xCoordFloor = (int)xCoord;
            if (xCoord < (double)xCoordFloor) {
                xCoordFloor--;
            }
            int xUnitCube = xCoordFloor & 0xFF;
            xCoord -= xCoordFloor;
            double fadeX = xCoord * xCoord * xCoord * (xCoord * (xCoord * 6.0 - 15.0) + 10.0);

            double zCoord = startZ + (double)zIteration * noiseFactorZ + octaveDatas[i].zOffset;
            int zCoordFloor = (int)zCoord;
            if (zCoord < (double)zCoordFloor) {
                zCoordFloor--;
            }
            int zUnitCube = zCoordFloor & 0xFF;
            zCoord -= zCoordFloor;
            double fadeZ = zCoord * zCoord * zCoord * (zCoord * (zCoord * 6.0 - 15.0) + 10.0);

            for (int yIteration = 0; yIteration < yIterationStart + yIterations; yIteration++) {
                double yCoord = startY + (double)yIteration * noiseFactorY + octaveDatas[i].yOffset;
                int yCoordFloor = (int)yCoord;
                if (yCoord < (double)yCoordFloor) {
                    yCoordFloor--;
                }
                int yUnitCube = yCoordFloor & 0xFF;
                yCoord -= yCoordFloor;
                double fadeY = yCoord * yCoord * yCoord * (yCoord * (yCoord * 6.0 - 15.0) + 10.0);

                if (yIteration == 0 || yUnitCube != i2) {
                    i2 = yUnitCube;
                    int j2 = getPermutation(octaveDatas[i].permutations, xUnitCube) + yUnitCube;
                    int k2 = getPermutation(octaveDatas[i].permutations, j2) + zUnitCube;
                    int l2 = getPermutation(octaveDatas[i].permutations, j2 + 1) + zUnitCube;
                    int i3 = getPermutation(octaveDatas[i].permutations, xUnitCube + 1) + yUnitCube;
                    int k3 = getPermutation(octaveDatas[i].permutations, i3) + zUnitCube;
                    int l3 = getPermutation(octaveDatas[i].permutations, i3 + 1) + zUnitCube;
                    d13 = lerp(fadeX, grad(getPermutation(octaveDatas[i].permutations, k2), xCoord, yCoord, zCoord), grad(getPermutation(octaveDatas[i].permutations, k3), xCoord - 1.0, yCoord, zCoord));
                    d15 = lerp(fadeX, grad(getPermutation(octaveDatas[i].permutations, l2), xCoord, yCoord - 1.0, zCoord), grad(getPermutation(octaveDatas[i].permutations, l3), xCoord - 1.0, yCoord - 1.0, zCoord));
                    d16 = lerp(fadeX, grad(getPermutation(octaveDatas[i].permutations, k2 + 1), xCoord, yCoord, zCoord - 1.0), grad(getPermutation(octaveDatas[i].permutations, k3 + 1), xCoord - 1.0, yCoord, zCoord - 1.0));
                    d18 = lerp(fadeX, grad(getPermutation(octaveDatas[i].permutations, l2 + 1), xCoord, yCoord - 1.0, zCoord - 1.0), grad(getPermutation(octaveDatas[i].permutations, l3 + 1), xCoord - 1.0, yCoord - 1.0, zCoord - 1.0));
                }
                double d28 = lerp(fadeY, d13, d15);
                double d29 = lerp(fadeY, d16, d18);
                double d30 = lerp(fadeZ, d28, d29);

                if (yIteration >= yIterationStart) {
                    CASTED_SHARED_MEMORY_ACCESS(yIteration - yIterationStart + sharedMemoryOffset) += d30 * octaveWidth;
                }
            }


            octavesFactor /= 2.0;
        }
    }


    __device__ void mixNoiseValues(int sharedMemoryOutputOffset, int sharedMemoryNoise1Offset, int sharedMemoryNoise2Offset, int sharedMemoryNoise3Offset, double noise6, int yAreaStart, int yAreas) {
        int i2 = 0;
        int j2 = 0;

        float f1 = 0.37000000476837158203125f;
        float f2 = -0.07500000298023223876953125;

        double d2 = noise6 / 8000.0;
        if (d2 < 0.0) {
            d2 = -d2 * 0.29999999999999999;
        }
        d2 = d2 * 3.0 - 2.0;
        if (d2 < 0.0) {
            d2 /= 2.0;
            if (d2 < -1.0) {
                d2 = -1.0;
            }
            d2 /= 1.3999999999999999;
            d2 /= 2.0;
        } else {
            if (d2 > 1.0) {
                d2 = 1.0;
            }
            d2 /= 8.0;
        }
        j2++;
        for (int k3 = yAreaStart; k3 < (yAreaStart + yAreas); k3++) {
            double d3 = f2;
            double d4 = f1;
            d3 += d2 * 0.20000000000000001;
            d3 = (d3 * (double)17) / 16.0;
            double d5 = (double)17 / 2.0 + d3 * 4.0;
            double d6 = 0.0;
            double d7 = (((double)k3 - d5) * 12.0 * 128.0) / (double)(1 << 7) / d4;
            if (d7 < 0.0) {
                d7 *= 4.0;
            }
            double d8 = CASTED_SHARED_MEMORY_ACCESS(i2 + sharedMemoryNoise1Offset) / 512.0;
            double d9 = CASTED_SHARED_MEMORY_ACCESS(i2 + sharedMemoryNoise2Offset) / 512.0;
            double d10 = (CASTED_SHARED_MEMORY_ACCESS(i2 + sharedMemoryNoise3Offset) / 10.0 + 1.0) / 2.0;

            if (d10 < 0.0) {
                d6 = d8;
            } else if (d10 > 1.0) {
                d6 = d9;
            } else {
                d6 = d8 + (d9 - d8) * d10;
            }

            d6 -= d7;
            if (k3 > 17 - 4) {
                double d11 = (float)(k3 - (17 - 4)) / 3.0f;
                d6 = d6 * (1.0 - d11) + -10.0 * d11;
            }
            CASTED_SHARED_MEMORY_ACCESS(i2 + sharedMemoryOutputOffset) = d6;
            i2++;
        }
    }

    __device__ void optimizedNoise(const NoiseData* __restrict__ noiseData, int sharedMemoryWriteOffset, int32_t x, int32_t y, int32_t z, int xArea, int zArea, int yAreaStart, int yAreas) {
        double noise6Value = optimizedNoise2D(noiseData->noise6, (double)x, (double)z, xArea, zArea, 200.0, 200.0, 16);

        for (int i = 0; i < yAreas; i++) {
            CASTED_SHARED_MEMORY_ACCESS(i) = 0.0;
        }
        for (int i = 0; i < yAreas; i++) {
            CASTED_SHARED_MEMORY_ACCESS(i + yAreas) = 0.0;
        }
        for (int i = 0; i < yAreas; i++) {
            CASTED_SHARED_MEMORY_ACCESS(i + yAreas + yAreas) = 0.0;
        }
        

        optimizedNoise3D(noiseData->noise1, 0, (double)x, (double)y, (double)z, xArea, zArea, 684.41200000000003, 684.41200000000003, 684.41200000000003, 16, yAreaStart, yAreas);
        optimizedNoise3D(noiseData->noise2, yAreas, (double)x, (double)y, (double)z, xArea, zArea, 684.41200000000003, 684.41200000000003, 684.41200000000003, 16, yAreaStart, yAreas);
        optimizedNoise3D(noiseData->noise3, yAreas + yAreas, (double)x, (double)y, (double)z, xArea, zArea, 8.5551500000000011, 4.2775750000000006, 8.5551500000000011, 8, yAreaStart, yAreas);
        mixNoiseValues(sharedMemoryWriteOffset, 0, yAreas, yAreas + yAreas, noise6Value, yAreaStart, yAreas);
    }

    __device__ void optimizedPointLerp(int sharedMemoryOffset, double bottomRight, double bottomLeft, double topRight, double topLeft, double bottomRight2, double bottomLeft2, double topRight2, double topLeft2, uint8_t baseHeight) {
        double bottomRightDiff = (bottomRight2 - bottomRight) * 0.125;
        double bottomLeftDiff = (bottomLeft2 - bottomLeft) * 0.125;
        double topRightDiff = (topRight2 - topRight) * 0.125;
        double topLeftDiff = (topLeft2 - topLeft) * 0.125;

        for (int y = 0; y < 8; y++) {
            double localBottomRight = bottomRight;
            double localTopRight = topRight;

            double localBottomRightDiff = (bottomLeft - bottomRight) * 0.25;
            double localTopRightDiff = (topLeft - topRight) * 0.25;

            for (int x = 0; x < 4; x++) {
                double localHeight = localBottomRight;
                double zStep = (localTopRight - localBottomRight) * 0.25;

                localHeight -= zStep;

                for (int z = 0; z < 4; z++) {
                    if ((localHeight += zStep) > 0.0) {
                        SHARED_MEMORY_ACCESS(x * 4 + z + sharedMemoryOffset) = baseHeight + y;
                    }
                }

                localBottomRight += localBottomRightDiff;
                localTopRight += localTopRightDiff;
            }

            bottomRight += bottomRightDiff;
            bottomLeft += bottomLeftDiff;
            topRight += topRightDiff;
            topLeft += topLeftDiff;
        }
    }

    __device__ uint8_t optimizedMod4Lerp(double a, double b, uint8_t baseHeight) {
        uint8_t height = 0;
        double diff = (b - a) * 0.125;
        for (int i = 0; i < 8; i++) {
            if (a > 0) {
                height = baseHeight + i;
            }
            a += diff;
        }

        return height;
    }
}


__device__ bool checkTerrain(uint64_t worldSeed) {
    Terrain::NoiseData noiseData;
    Terrain::initializeNoise(worldSeed, &noiseData);

    Terrain::optimizedNoise(&noiseData, 9, -22 * 4, 0, 2 * 4, 0, 2, 8, 2);
    if (Terrain::optimizedMod4Lerp(CASTED_SHARED_MEMORY_ACCESS(9), CASTED_SHARED_MEMORY_ACCESS(10), 64) != 65) { 
        return false;
    }

    Terrain::optimizedNoise(&noiseData, 11, -22 * 4, 0, 2 * 4, 1, 2, 8, 2);
    if (Terrain::optimizedMod4Lerp(CASTED_SHARED_MEMORY_ACCESS(11), CASTED_SHARED_MEMORY_ACCESS(12), 64) != 67) { 
        return false;
    }

    Terrain::optimizedNoise(&noiseData, 13, -22 * 4, 0, 2 * 4, 0, 3, 8, 2);
    if (Terrain::optimizedMod4Lerp(CASTED_SHARED_MEMORY_ACCESS(13), CASTED_SHARED_MEMORY_ACCESS(14), 64) != 67) { 
        return false;
    }

    Terrain::optimizedNoise(&noiseData, 15, -22 * 4, 0, 2 * 4, 1, 3, 7, 3);
    if (CASTED_SHARED_MEMORY_ACCESS(16) > 0) { return false; }
    if (Terrain::optimizedMod4Lerp(CASTED_SHARED_MEMORY_ACCESS(15), CASTED_SHARED_MEMORY_ACCESS(16), 56) != 63) { 
        return false;
    }
    
    Terrain::optimizedNoise(&noiseData, 18, -22 * 4, 0, 2 * 4, 2, 3, 7, 2);
    if (CASTED_SHARED_MEMORY_ACCESS(19) > 0) { return false; }
    if (Terrain::optimizedMod4Lerp(CASTED_SHARED_MEMORY_ACCESS(18), CASTED_SHARED_MEMORY_ACCESS(19), 56) != 63) { 
        return false;
    }


    
    int sharedMemoryOffset = 0;
    for (int i = 0; i < 16; i++) {
        SHARED_MEMORY_ACCESS(sharedMemoryOffset + i) = 0; 
    }
    Terrain::optimizedPointLerp(sharedMemoryOffset, CASTED_SHARED_MEMORY_ACCESS(9), CASTED_SHARED_MEMORY_ACCESS(11), CASTED_SHARED_MEMORY_ACCESS(13), CASTED_SHARED_MEMORY_ACCESS(16), CASTED_SHARED_MEMORY_ACCESS(10), CASTED_SHARED_MEMORY_ACCESS(12), CASTED_SHARED_MEMORY_ACCESS(14), CASTED_SHARED_MEMORY_ACCESS(17), 64);

    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 2) != 66) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 3) != 67) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 4) != 65) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 6) != 66) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 7) != 66) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 8) != 65) { return false; }
    // if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 9) != 65) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 12) != 66) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 13) != 65) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 14) != 64) { return false; }
    if (SHARED_MEMORY_ACCESS(sharedMemoryOffset + 15) != 64) { return false; }

    return true;
}


__device__ __managed__ uint32_t outputCounter = 0;
__device__ __managed__ uint64_t outputBuffer[100000];
__global__ void __launch_bounds__(THREAD_COUNT, 3) gpuWork(uint64_t seedOffset) {
    uint64_t worldSeed = (uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x + seedOffset;

    if (!checkTerrain(worldSeed)) {
        return;
    }

    uint32_t idx = atomicAdd(&outputCounter, 1);
    outputBuffer[idx] = worldSeed;
}


uint64_t milliseconds() {
    return (std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())).count();
}

#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
        boinc_finish(code);
    }
}


int calculateBlockSize(double threshold) {
    gpuWork<<<1, THREAD_COUNT>>>(0);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    GPU_ASSERT(hipPeekAtLastError());
    outputCounter = 0;

    int setBits = 0;
    int lowestSetBit = 30;
    for (int i = 0; i < 30; i++) {
        int j;
        for (j = 0; j < lowestSetBit; j++) {
            int32_t newBits = setBits | (1 << j);

            uint64_t startTime = milliseconds();

            gpuWork<<<newBits, THREAD_COUNT>>>(0);
            GPU_ASSERT(hipPeekAtLastError());
            GPU_ASSERT(hipDeviceSynchronize());
            GPU_ASSERT(hipPeekAtLastError());
            outputCounter = 0;

            uint64_t endTime = milliseconds();

            double elapsed = (double)(endTime - startTime) / 1000.0;
            
            if (elapsed > threshold) {
                if (j != 0) {
                    setBits |= (1 << (j - 1));
                    lowestSetBit = (j - 1);
                } else if (j == 0) {
                    lowestSetBit = 0;
                }
                break;
            }
        }

        if (lowestSetBit == 0) { break; }

        if (j == lowestSetBit) {
            setBits |= (1 << (j - 1));
            lowestSetBit = (j - 1);
        }
    }

    return setBits;
}


struct CheckpointData {
    int lastIteration;
    double elapsed;
    int blockCount;
};

int main(int argc, char* argv[]) {
    int taskNumber = 0;
    int device = 0;
    for (int i = 1; i < argc; i += 2) {
        const char *param = argv[i];
        if (strcmp(param, "-t") == 0 || strcmp(param, "--task") == 0) {
            taskNumber = atoi(argv[i + 1]);
        } else if (strcmp(param, "-d") == 0 || strcmp(param, "--device") == 0) {
            device = atoi(argv[i + 1]);
        }
    }

    int startIteration = 0;
    double elapsed = 0;
    int BLOCK_COUNT = 0;

    fprintf(stderr, "Recieved work unit: %d.\n", taskNumber);
    fflush(stderr);

    #ifdef BOINC
        BOINC_OPTIONS options;
        boinc_options_defaults(options);
        options.normal_thread_priority = true; 
        boinc_init_options(&options);

        APP_INIT_DATA aid;
        boinc_get_init_data(aid);
        
        if (aid.gpu_device_num >= 0) {
            fprintf(stderr, "boinc gpu: %d, cli gpu: %d.\n", aid.gpu_device_num, device);
            device = aid.gpu_device_num;
        } else {
            fprintf(stderr, "cli gpu: %d.\n", device);
        }
    #endif

    hipSetDevice(device);
    GPU_ASSERT(hipPeekAtLastError());
    GPU_ASSERT(hipDeviceSynchronize());
    GPU_ASSERT(hipPeekAtLastError());

    FILE* checkpointFile = boinc_fopen("trailer_checkpoint.txt", "rb");

    if (checkpointFile) {
        boinc_begin_critical_section();
        struct CheckpointData checkpointData;

        fread(&checkpointData, sizeof(checkpointData), 1, checkpointFile);
        startIteration = checkpointData.lastIteration + 1;
        elapsed = checkpointData.elapsed;
        BLOCK_COUNT = checkpointData.blockCount;

        fclose(checkpointFile);
        fprintf(stderr, "Loaded checkpoint %d %.2f %d.\n", startIteration, elapsed, BLOCK_COUNT);
        fflush(stderr);
        boinc_end_critical_section();
    } else {
        fprintf(stderr, "No checkpoint to load.\n");
    }
    if (BLOCK_COUNT == 0) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        int cc = deviceProp.major * 10 + deviceProp.minor;

        if (cc <= 52) {
            BLOCK_COUNT = calculateBlockSize(0.02);
        } else if (deviceProp.major == 6) {
            BLOCK_COUNT = calculateBlockSize(0.1);
        } else if (deviceProp.major == 7) {
            BLOCK_COUNT = calculateBlockSize(0.15);
        } else if (deviceProp.major == 8) {
            BLOCK_COUNT = calculateBlockSize(0.5);
        } else {
            fprintf(stderr, "Unrecognized compute capability.\n");
            fflush(stderr);
            boinc_finish(1);
        }
        fprintf(stderr, "Calculated block count: %d.\n", BLOCK_COUNT);
        if (BLOCK_COUNT == 0) { BLOCK_COUNT = 1; }
        fflush(stderr);
    }

    uint64_t GRID_WORK = (uint64_t)BLOCK_COUNT * THREAD_COUNT;
    int ITERATIONS_NEEDED = ((TASK_WORK + GRID_WORK - 1) / GRID_WORK);

    for (int i = startIteration; i < ITERATIONS_NEEDED; i++) {
        uint64_t seedOffset = (TASK_WORK * taskNumber) + GRID_WORK * i;
        uint64_t startTime = milliseconds();

        gpuWork<<<BLOCK_COUNT, THREAD_COUNT>>>(seedOffset);
        GPU_ASSERT(hipPeekAtLastError());
        GPU_ASSERT(hipDeviceSynchronize());
        GPU_ASSERT(hipPeekAtLastError());

        uint64_t endTime = milliseconds();

        boinc_begin_critical_section();

        double localElapsed = ((double)(endTime - startTime) / 1000);
        elapsed += localElapsed;

        if (boinc_time_to_checkpoint()) {
            struct CheckpointData checkpointData;
            checkpointData.lastIteration = i;
            checkpointData.elapsed = elapsed;
            checkpointData.blockCount = BLOCK_COUNT;

            FILE* checkpointFile = boinc_fopen("trailer_checkpoint.txt", "wb");
            fwrite(&checkpointData, sizeof(checkpointData), 1, checkpointFile);
            fclose(checkpointFile);

            boinc_checkpoint_completed();
        }

        if (outputCounter > 0) {
            FILE *seedsOut = boinc_fopen("trailer_seeds.txt", "a");
            for (int j = 0; j < outputCounter; j++) {
                if (outputBuffer[j] < (TASK_WORK * (taskNumber + 1))) {
                    fprintf(seedsOut, "Seed: %llu\n", outputBuffer[j]);
                }
            }
            fclose(seedsOut);
            outputCounter = 0;
        }

        double fracDone = (double)i / ITERATIONS_NEEDED;
        boinc_fraction_done(fracDone);

        boinc_end_critical_section();   
    }

    boinc_begin_critical_section();
    FILE *seedsOut = boinc_fopen("trailer_seeds.txt", "a");
    fclose(seedsOut);

    fprintf(stderr, "Finished in %.2f seconds. Speed: %.2f/s.\n", elapsed, (double)TASK_WORK / elapsed);
    fflush(stderr);
    boinc_delete_file("trailer_checkpoint.txt");

    boinc_end_critical_section();

    boinc_finish(0);
}